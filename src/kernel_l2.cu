#include "hip/hip_runtime.h"

// Should be multiple of 32
// #define L2_CACHE_SIZE 98304 //96KB
// #define L2_CACHE_SIZE 196608 // 192KB 
// Should be multiple of 32 and power of 2
#define CACHED_ARRAY_SIZE  65536   // 64KB
#define BLOCK_SIZE     1024
static __device__ char arr[CACHED_ARRAY_SIZE];

// Referred to code in https://arxiv.org/pdf/1804.06826.pdf
static __global__ void l2_ld_hammer_kernel()
{
    constexpr int ntmo = BLOCK_SIZE - 1;
    constexpr int nd = CACHED_ARRAY_SIZE / 8;
    double x = 0;
    int tid = threadIdx.x;
    for (int it = 0; it < 12000000; ++it) {
        double *ptr = (double *)arr;
        for (int i = 0; i < nd; i += BLOCK_SIZE) {
            #pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j += 32) {
                int offset = (tid + j) & ntmo;
                asm volatile ("{\t\n"
                    ".reg .f64 val;\n\t"
                    "ld.global.cg.f64 val, [%1];\n\t"
                    "add.f64 %0, val, %0;\n\t"
                    "}" : "+d"(x) : "l"(ptr+offset) : "memory"
                );
            }
            ptr += 32;
        }
    }
    // For avoiding compiler optimization.
    ((double *)arr)[tid] = x;
}

extern "C" {

hipError_t l2_ld_hammer(hipStream_t s, int nblks)
{
    dim3 grid = dim3(nblks, 1, 1);
    dim3 block = dim3(BLOCK_SIZE, 1, 1);
    return hipLaunchKernel(reinterpret_cast<const void*>((void *)l2_ld_hammer_kernel), grid, block, 0, 0, s);
}

} // extern "C"

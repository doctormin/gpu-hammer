#include "hip/hip_runtime.h"

// Should be multiple of 32
#define ARRAY_SIZE      268435456   // 256M
// Should be multiple of 32 and power of 2
#define BLOCK_SIZE     640
//
#define UNROLL_DEPTH    128

static __device__ char ld_arr[ARRAY_SIZE];

static __global__ void gmem_ld_hammer_kernel()
{
    int nd = ARRAY_SIZE >> 3;
    volatile double *ptr = (volatile double *)ld_arr;
    double x = 0;
    int idx = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    int sz = BLOCK_SIZE * gridDim.x;
    int usz = sz * UNROLL_DEPTH;
    nd -= (nd % usz);
    for (int it = 0; it < 100000; ++it) {
        for (int i = idx; i < nd; i += usz) {
            #pragma unroll
            for (int j = 0; j < UNROLL_DEPTH; ++j) {
                x += ptr[i+j*sz];
            }
        }
    }
    // For avoiding compiler optimization.
    ((double *)ptr)[idx] = x;
}

extern "C" {

hipError_t gmem_ld_hammer(hipStream_t s, int nblks)
{
    dim3 grid = dim3(nblks, 1, 1);
    dim3 block = dim3(BLOCK_SIZE, 1, 1);
    return hipLaunchKernel(reinterpret_cast<const void*>((void *)gmem_ld_hammer_kernel), grid, block, 0, 0, s);
}

} // extern "C"
